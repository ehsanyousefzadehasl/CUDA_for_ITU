#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void squareArrayGPU(int *d_input, int *d_output, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        d_output[idx] = d_input[idx] * d_input[idx];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);
    if (size <= 0) {
        printf("Array size must be a positive integer.\n");
        return 1;
    }

    int *input = (int *)malloc(size * sizeof(int));
    int *output = (int *)malloc(size * sizeof(int));
    int *d_input, *d_output;
    int bytes = size * sizeof(int);

    for (int i = 0; i < size; i++) {
        input[i] = i + 1;
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipMalloc((void **)&d_input, bytes);
    hipMalloc((void **)&d_output, bytes);

    hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    

    // Start GPU timing
    hipEventRecord(start);

    // Launch the kernel
    squareArrayGPU<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);
    hipDeviceSynchronize();
    

    hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);



    // Stop GPU timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);

    printf("GPU Kernel Execution Time: %.6f ms\n", gpu_time);


    hipFree(d_input);
    hipFree(d_output);
    free(input);
    free(output);

    return 0;
}
