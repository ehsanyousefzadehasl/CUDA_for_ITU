#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32  // Number of threads per block
#define RADIUS 3        // Stencil radius
#define ARRAY_SIZE 100   // Input array size (for demonstration)

// CUDA kernel: 1D Stencil
__global__ void stencil_1d(int *in, int *out, int n) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];  // Shared memory with halo

    int gindex = threadIdx.x + blockIdx.x * blockDim.x;  // Global index
    int lindex = threadIdx.x + RADIUS;                  // Local index in shared memory

    // Load input elements into shared memory
    temp[lindex] = (gindex < n) ? in[gindex] : 0;

    // Load the halo (left and right)
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = (gindex >= RADIUS) ? in[gindex - RADIUS] : 0;
        temp[lindex + BLOCK_SIZE] = (gindex + BLOCK_SIZE < n) ? in[gindex + BLOCK_SIZE] : 0;
    }

    // Synchronize (ensure all the data is available before computing)
    __syncthreads();

    // Apply the stencil
    if (gindex < n) {  // Ensure we're within bounds
        int result = 0;
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            result += temp[lindex + offset];
        }
        // Store the result in the output array
        out[gindex] = result;
    }
}

int main() {
    // Input and output array sizes
    int n = ARRAY_SIZE;

    // Host memory allocation
    int *h_in = (int *)malloc(n * sizeof(int));
    int *h_out = (int *)malloc(n * sizeof(int));

    // Initialize the input array with some values
    printf("Input array:\n");
    for (int i = 0; i < n; i++) {
        h_in[i] = i + 1;  // Example: [1, 2, 3, ..., n]
        printf("%d ", h_in[i]);
    }
    printf("\n");

    // Device memory allocation
    int *d_in, *d_out;
    hipMalloc((void **)&d_in, n * sizeof(int));
    hipMalloc((void **)&d_out, n * sizeof(int));

    // Copy input array to device
    hipMemcpy(d_in, h_in, n * sizeof(int), hipMemcpyHostToDevice);

    // Configure kernel launch parameters
    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the stencil kernel
    stencil_1d<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, n);

    // Copy result back to host
    hipMemcpy(h_out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the output array
    printf("Output array:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;
}
