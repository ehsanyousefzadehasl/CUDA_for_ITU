#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16
#define FILTER_RADIUS 1

__constant__ int SOBEL_X[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int SOBEL_Y[3][3] = {
    {-1, -2, -1},
    {0,  0,  0},
    {1,  2,  1}
};

__global__ void sobelEdgeDetection(const unsigned char *input, unsigned char *output, int width, int height) {
    __shared__ unsigned char tile[TILE_SIZE + 2 * FILTER_RADIUS][TILE_SIZE + 2 * FILTER_RADIUS];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + tx - FILTER_RADIUS;
    int row = blockIdx.y * TILE_SIZE + ty - FILTER_RADIUS;

    if (row >= 0 && row < height && col >= 0 && col < width) {
        tile[ty][tx] = input[row * width + col];
    } else {
        tile[ty][tx] = 0;
    }
    __syncthreads();

    if (tx >= FILTER_RADIUS && tx < TILE_SIZE + FILTER_RADIUS &&
        ty >= FILTER_RADIUS && ty < TILE_SIZE + FILTER_RADIUS &&
        row < height && col < width) {
        int Gx = 0, Gy = 0;

        for (int i = -FILTER_RADIUS; i <= FILTER_RADIUS; i++) {
            for (int j = -FILTER_RADIUS; j <= FILTER_RADIUS; j++) {
                Gx += SOBEL_X[i + FILTER_RADIUS][j + FILTER_RADIUS] * tile[ty + i][tx + j];
                Gy += SOBEL_Y[i + FILTER_RADIUS][j + FILTER_RADIUS] * tile[ty + i][tx + j];
            }
        }

        int magnitude = abs(Gx) + abs(Gy);
        output[row * width + col] = (magnitude > 255) ? 255 : magnitude;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_image> <output_image>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char *h_input = stbi_load(argv[1], &width, &height, &channels, 1); // Load as grayscale
    if (!h_input) {
        printf("Failed to load image: %s\n", argv[1]);
        return 1;
    }

    int imageSize = width * height;
    unsigned char *h_output = (unsigned char *)malloc(imageSize * sizeof(unsigned char));

    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, imageSize * sizeof(unsigned char));
    hipMalloc((void **)&d_output, imageSize * sizeof(unsigned char));

    hipMemcpy(d_input, h_input, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE + 2 * FILTER_RADIUS, TILE_SIZE + 2 * FILTER_RADIUS);
    dim3 blocksPerGrid((width + TILE_SIZE - 1) / TILE_SIZE, (height + TILE_SIZE - 1) / TILE_SIZE);

    sobelEdgeDetection<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, width, height);

    hipMemcpy(h_output, d_output, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    stbi_write_png(argv[2], width, height, 1, h_output, width); // Save result as PNG

    printf("Edge detection complete. Result saved to: %s\n", argv[2]);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
